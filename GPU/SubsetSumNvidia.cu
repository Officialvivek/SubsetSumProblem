// Saniyah Bokhari
// April 15, 2011
// SubsetSumNvidia.cu; starts from test10.cu
//            fresh blocking code
// nvcc  -DPRINTIT -DPRINTTIME -DPRINTSUBSET -DPRINTTABLE -o SubsetSumNvidia SubsetSumNvidia.cu



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <stdint.h>
#include <sys/time.h>
#define UINT unsigned
#ifdef OSC
#define mainMemory 2.14E10
#define deviceMemory 3.0E9
#else
#define mainMemory 2.0E9
#define deviceMemory 3.0E8
#endif

void checkCUDAError(const char *msg) {
   hipError_t err = hipGetLastError();
   if( hipSuccess != err) {
      fprintf(stderr, "Cuda error: %s: %s.\n", msg,
                       hipGetErrorString( err));
      exit(EXIT_FAILURE);
   }
}
#define MIN(x,y)  ((x)<(y)?(x):(y))

__device__ __constant__ int valsDev[16384];//put in constant memory

int tstBitInArray(UINT d[], int numWords, int index);

__global__ void kernel0(UINT* devPtr){
   int tid = blockIdx.x * blockDim.x + threadIdx.x;
   UINT* row = (UINT*)((char*)devPtr);
   if(tid==0)
      row[tid]=1U;
   else
      row[tid]=0U;
}

__global__ void kernel1(UINT* devPtr, UINT* devPtr2, int pitch, int i){
   UINT* row = (UINT*)((char*)devPtr + i * pitch);
   UINT* row2;
   if(i==0)
      row2 = devPtr2;
   else
      row2 = (UINT*)((char*)devPtr + (i-1) * pitch);
   int tid = blockDim.x * blockIdx.x + threadIdx.x;
   row[tid] = row2[tid];
}

__global__ void kernel2(UINT* devPtr, UINT* devPtr2, int pitch, int width, UINT
halfSize, int i, int si, int start){
   int bit, wNew;
   UINT upper, temp11, temp12, temp21,temp22;
   int size = valsDev[si];
   UINT* row = (UINT*)((char*)devPtr + i * pitch);
   UINT* row2;
   if(i==0)
      row2 = devPtr2;
   else
      row2 = (UINT*)((char*)devPtr + (i-1) * pitch);
   int w = blockDim.x * blockIdx.x + threadIdx.x;
   if(((w & 1)==start)&&(w<width+1)){
      bit = (size % 32);
      wNew = w + (size/32);
      upper=row2[w];
      if(((w+1)*32+size)<=(halfSize+31)){
         temp11=row[wNew];
         temp12=temp11|((upper)<<bit);
         row[wNew]=temp12;
         if(bit!=0){
            temp21 = row[wNew+1];
            temp22 = temp21|((upper)>>(32-bit));
            row[wNew+1]=temp22;
         }
      }   
   }
}

int main(int argc, char *argv[])
{
   int *vals;
   UINT *d; //linearized array to store binary 1's and 0's
   UINT *dDev;
   UINT *eDev;
   int *ans;
   
   int numObj, seed;
   int numObjLo, numObjHi, sizeLo, sizeHi;
   int count = 0;
   int runs;

   assert(argc==8);
   numObjLo = atoi(argv[1]); 
   numObjHi = atoi(argv[2]);
   sizeLo = atoi(argv[3]);
   sizeHi = atoi(argv[4]);
   seed = atoi(argv[5]);
   int block_size = atoi(argv[6]);
   runs = atoi(argv[7]);

   srand(seed); //seed random number generator

   struct timeval tvTab1, tvTab2,
                  tvFree1, tvFree2, tvBac1, tvBac2;
   long tTab, tFree, tBac;
   double secTab, secFree, secBac, secPer;

do{
  
   UINT sumSize = 0;
   int k;
   numObj = numObjLo+(int)((numObjHi-numObjLo)*((double)rand()/(double)(RAND_MAX)));
   
   #ifdef PRINTIT
      printf("(n %d %d %d)(s %d)\n",
      numObjLo, numObjHi, numObj, seed);
   #endif

   vals = (int *)malloc((1+numObj)*sizeof(int));  
   assert(vals!=NULL);
   ans = (int *)malloc((1+numObj)*sizeof(int)); 
   assert(ans!=NULL);
   
   int rows=1+numObj;

   vals[0] = 0;
      
   #ifdef PRINTIT
      printf("the problem:\n");
   #endif
          
   for(k=1;k<=numObj;k++){
      vals[k]= sizeLo+(int)((sizeHi-sizeLo)*(((double)rand()/(double)(RAND_MAX))));
      #ifdef PRINTIT
      printf("%d  ",vals[k]);
      #endif
      sumSize = sumSize + vals[k];
   }    
   #ifdef PRINTIT
      printf("\n");
   #endif
  
   UINT halfSize = sumSize/2; 
   int i,j;

   int words; //number of words needed
   words = (int)(ceil(((double)(halfSize))/32.0));
   
   
   //If sumSize is Odd then no subset is possible

   if((sumSize % 2)!= 0){
      #ifdef PRINTTABLE
      printf(" -- No subset was Found as Sumsize is odd -- \n");
      #endif
      free(vals);
      free(ans);
      continue;
   }
   double warea=(double)words*(double)numObj;
   double thebytes=(double)(sizeof(UINT))*warea;
   double gig = thebytes/(double)(1.073E9);
   printf("words %d numObj %d w-area %10lg bytes %lg gig %lg\n",words,numObj,warea,thebytes,gig); 
   if(thebytes > mainMemory){
      fprintf(stderr," -- Aborted, required main Memory %lg  > %lg -- \n",thebytes,mainMemory);fflush(stderr);
      free(vals);
      free(ans);
      exit(1);
   }

   int num_elements = (words+1);
   int grid_size = (int)ceil((double)(num_elements) / (double)block_size);
   if(grid_size>65535){
      fprintf(stderr,"problem too large for grid\n");fflush(stderr);
      exit(1);
   }

   d = (UINT *)malloc((1+numObj)*(words+1)*sizeof(UINT));  
   assert(d!=NULL);
   
   size_t pitch;
   
   int rowsPerChunk = MIN((int)ceil(deviceMemory/(sizeof(UINT)*(double)(words+1))),(1+numObj));
   int chunks = (int)ceil((double)rows/(double)rowsPerChunk);
   fprintf(stderr,"rows = %d ",rows);
   fprintf(stderr," chunks = %d  rowsPerChunk = %d\n",chunks, rowsPerChunk);
   gettimeofday(&tvTab1, NULL);
   hipMallocPitch((void**)&dDev,&pitch,(words+1)*sizeof(UINT),rowsPerChunk);  
   assert(dDev!=NULL);
   hipMalloc((void**)&eDev,pitch);  
   assert(eDev!=NULL);

   hipMemcpyToSymbol(HIP_SYMBOL("valsDev"), vals, rows*sizeof(int));
   int ch, startRow, endRow;
   #ifdef PRINTERROR
   int num_bytes = num_elements * sizeof(UINT);
   printf("num_elements (words+1) = %d (%lu bytes)",num_elements,num_elements*sizeof(UINT));
   printf("pitch = %lu ",pitch);
   printf("numObj = %d\n",numObj);
   printf("d is = %d (%lg MB) ",(1+numObj)*(words+1),(double)((1+numObj)*(words+1)*sizeof(UINT))/1000000.0);
   printf("dDev is = %lu (%lg MB)\n",(pitch)*(1+numObj),(double)((pitch)*(1+numObj))/1000000.0);
   printf("num_bytes = %d, ",num_bytes);
   printf("grid_size %d, ",grid_size);
   printf("block_size %d\n",block_size);
   for(ch=0;ch<chunks;ch++){
      startRow=ch*rowsPerChunk;
      endRow=MIN(((ch+1)*rowsPerChunk),rows);
      printf("chunk %d, row %d to row %d\n", ch, startRow, endRow);
   }
   #endif
 
   int thisChunkSize;
   gettimeofday(&tvTab1, NULL);
   kernel0<<<grid_size,block_size>>>(dDev); //init first row
   for(ch=0;ch<chunks;ch++){                  //for each chunk
      startRow=ch*rowsPerChunk;               //starting row == 0th chunk row
      endRow=MIN(((ch+1)*rowsPerChunk),rows); //ending row == last chunk row + 1
      thisChunkSize = endRow - startRow;
      for(i=0; i<thisChunkSize; i++){     //iterate through chunk
         if(((ch==0)&&(i==0))){ //don't do this for 0th row of 0th chunk
            //printf("ch = %d i = %d ; do nothing\n", ch, i);
         }
         else{
            int start;
            int si=ch*rowsPerChunk+i;//the real index
            kernel1<<<grid_size,block_size>>>(dDev, eDev, pitch, i);//copy previous row
            for(start=0;start<2;start++){
               kernel2<<<grid_size,block_size>>>(dDev, eDev, pitch, words+1, halfSize, i, si, start);
            } 
         }
      }
      int lastRowInChunk = rowsPerChunk;
      //this is incorrect for last row in last chunk, but that is never used anyway
      hipMemcpy(eDev, dDev+(lastRowInChunk-1)*(pitch/sizeof(UINT)), pitch, hipMemcpyDeviceToDevice);
      int rowsToTransfer = thisChunkSize;
      hipMemcpy2D(d+ch*rowsPerChunk*(words+1), (words+1)*sizeof(UINT),
                   dDev, pitch,
                   (words+1)*sizeof(UINT),rowsToTransfer,hipMemcpyDeviceToHost);
                  //copy this chunk out to host in the appropriate location
   }
   gettimeofday(&tvTab2, NULL);


    if((tstBitInArray(&d[numObj*(words+1)],words,halfSize))!= 1){
       #ifdef PRINTTABLE
       printf("++ No subset was Found as last element != halfSize ++\n");
       #endif
    }    
    
    gettimeofday(&tvBac1, NULL);
    int testSum = 0;
    int index = 0;
    i = numObj;
    j = halfSize;
    int j2;
    if((tstBitInArray(&d[i*(words+1)],words,j))== 1){
       for(i=numObj;i>0;i--){
          if(((tstBitInArray(&d[(i-1)*(words+1)],words,j))== 1)&&((tstBitInArray(&d[i*(words+1)],words,j))== 1)){
             //go up
          }
          else{
            j2 = j;
            int diff;
            diff = j2-vals[i];
            j = diff;
            ans[index] = vals[i];
            //break;
            testSum +=ans[index];
            #ifdef PRINTSUBSET  
            printf(" %2d %2d %u %2d\n",ans[index],vals[i],halfSize,testSum);
            #endif
            index++;
         }
      }
    }
    #ifdef PRINTSUBSET 
    if(testSum == halfSize)
      printf("Sum Correct\n");
    else
       printf("Sum incorrect\n");
    #endif
    gettimeofday(&tvBac2, NULL);


    #ifdef PRINTTABLE
    UINT sumSize2 = 0; 
    printf("  S   W    ");  
    for(j=0;j<=halfSize;j++){
       printf(" %2d",j%100);
    }
    printf("\n");
    for(i=0;i<=numObj;i++){
       sumSize2 = sumSize2 + vals[i];
       printf("%3u %3d  %2d ",sumSize2,vals[i],i);
       for(j=0;j<=halfSize;j++){
          if((tstBitInArray(&d[i*(words+1)],words,j))== 1){
             printf(" 1 ");
          }
          if((tstBitInArray(&d[i*(words+1)],words,j))== 0){
             printf(" 0 ");
          }
       }
       printf("\n");
    }
    #endif
    
    gettimeofday(&tvFree1, NULL);
    free(vals);
    free(ans);
    free(d);
    hipFree(dDev);
    hipFree(eDev);
    gettimeofday(&tvFree2, NULL);

      #ifdef PRINTTIME

      //Timing for creating the table
      tTab = (tvTab2.tv_sec - tvTab1.tv_sec)*1000000 + tvTab2.tv_usec - tvTab1.tv_usec;
      secTab = (double)tTab/1000000.0;

      //Timing for backtracking
      tBac = (tvBac2.tv_sec - tvBac1.tv_sec)*1000000 + tvBac2.tv_usec - tvBac1.tv_usec;
      secBac = (double)tBac/1000000.0;

     //Timing for freeing
      tFree = (tvFree2.tv_sec - tvFree1.tv_sec)*1000000 + tvFree2.tv_usec - tvFree1.tv_usec;
      secFree = (double)tFree/1000000.0;

     unsigned long bitarea = (unsigned long)numObj*(unsigned long)halfSize;
      secPer=(secTab)/(double)(warea);
      printf("%3d Ob %4d w %7d a %10lg gig %5lg ba %10lu ",
             count,numObj,words,warea,gig,bitarea);
      printf("sTab %8g sBac %7g sFree %10g sPer %10g N",
             secTab,secBac,secFree,secPer);
      printf("\n");fflush(stdout);
      #endif

    count++;
 
    }while (count < runs);

    return 0;

}

int tstBitInArray(UINT d[], int numWords, int index){
   //   in array d[numWords] return the value of bit index
   int bit;
   bit = index % 32;
   int word;
   word = index/32;
   UINT t=(1U<<bit);
   if((d[word] & t)>0)
      return(1);
   else
      return(0);    
}
